#include "hip/hip_runtime.h"
#include "common.h"
#include "reduce.h"
#if CUDART_VERSION >= 11080
  #if __CUDA_ARCH__ < 900
    DEFINE_ncclDevKernel_nop(Reduce_Sum_f8e4m3_RING_LL, ncclFuncReduce, FuncSum, __hip_fp8_e4m3_fnuz, NCCL_ALGO_RING, NCCL_PROTO_LL, 438)
  #else
    DEFINE_ncclDevKernel(Reduce_Sum_f8e4m3_RING_LL, ncclFuncReduce, FuncSum, __hip_fp8_e4m3_fnuz, NCCL_ALGO_RING, NCCL_PROTO_LL, 438)
  #endif
#endif
#if CUDART_VERSION >= 11080 && __CUDA_ARCH__ >= 900
DEFINE_ncclDevFunc(Reduce_Sum_f8e4m3_RING_LL, ncclFuncReduce, FuncSum, __hip_fp8_e4m3_fnuz, NCCL_ALGO_RING, NCCL_PROTO_LL)
#endif
#if CUDART_VERSION >= 11080 && __CUDA_ARCH__ >= 900
DEFINE_ncclDevFunc(Reduce_Sum_f8e4m3_RING_LL128, ncclFuncReduce, FuncSum, __hip_fp8_e4m3_fnuz, NCCL_ALGO_RING, NCCL_PROTO_LL128)
#endif
#if CUDART_VERSION >= 11080 && __CUDA_ARCH__ >= 900
DEFINE_ncclDevFunc(Reduce_Sum_f8e4m3_RING_SIMPLE, ncclFuncReduce, FuncSum, __hip_fp8_e4m3_fnuz, NCCL_ALGO_RING, NCCL_PROTO_SIMPLE)
#endif
