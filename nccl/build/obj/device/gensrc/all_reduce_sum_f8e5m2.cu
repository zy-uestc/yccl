#include "hip/hip_runtime.h"
#include "common.h"
#include "all_reduce.h"
#if CUDART_VERSION >= 11080
  #if __CUDA_ARCH__ < 900
    DEFINE_ncclDevKernel_nop(AllReduce_Sum_f8e5m2_RING_LL, ncclFuncAllReduce, FuncSum, __hip_fp8_e5m2_fnuz, NCCL_ALGO_RING, NCCL_PROTO_LL, 284)
  #else
    DEFINE_ncclDevKernel(AllReduce_Sum_f8e5m2_RING_LL, ncclFuncAllReduce, FuncSum, __hip_fp8_e5m2_fnuz, NCCL_ALGO_RING, NCCL_PROTO_LL, 284)
  #endif
#endif
#if CUDART_VERSION >= 11080
  #if __CUDA_ARCH__ < 900
    DEFINE_ncclDevKernel_nop(AllReduce_Sum_f8e5m2_TREE_LL, ncclFuncAllReduce, FuncSum, __hip_fp8_e5m2_fnuz, NCCL_ALGO_TREE, NCCL_PROTO_LL, 287)
  #else
    DEFINE_ncclDevKernel(AllReduce_Sum_f8e5m2_TREE_LL, ncclFuncAllReduce, FuncSum, __hip_fp8_e5m2_fnuz, NCCL_ALGO_TREE, NCCL_PROTO_LL, 287)
  #endif
#endif
#if CUDART_VERSION >= 11080 && __CUDA_ARCH__ >= 900
DEFINE_ncclDevFunc(AllReduce_Sum_f8e5m2_COLLNET_CHAIN_SIMPLE, ncclFuncAllReduce, FuncSum, __hip_fp8_e5m2_fnuz, NCCL_ALGO_COLLNET_CHAIN, NCCL_PROTO_SIMPLE)
#endif
#if CUDART_VERSION >= 11080 && __CUDA_ARCH__ >= 900
DEFINE_ncclDevFunc(AllReduce_Sum_f8e5m2_COLLNET_DIRECT_SIMPLE, ncclFuncAllReduce, FuncSum, __hip_fp8_e5m2_fnuz, NCCL_ALGO_COLLNET_DIRECT, NCCL_PROTO_SIMPLE)
#endif
#if CUDART_VERSION >= 11080 && __CUDA_ARCH__ >= 900
DEFINE_ncclDevFunc(AllReduce_Sum_f8e5m2_RING_LL, ncclFuncAllReduce, FuncSum, __hip_fp8_e5m2_fnuz, NCCL_ALGO_RING, NCCL_PROTO_LL)
#endif
#if CUDART_VERSION >= 11080 && __CUDA_ARCH__ >= 900
DEFINE_ncclDevFunc(AllReduce_Sum_f8e5m2_RING_LL128, ncclFuncAllReduce, FuncSum, __hip_fp8_e5m2_fnuz, NCCL_ALGO_RING, NCCL_PROTO_LL128)
#endif
#if CUDART_VERSION >= 11080 && __CUDA_ARCH__ >= 900
DEFINE_ncclDevFunc(AllReduce_Sum_f8e5m2_RING_SIMPLE, ncclFuncAllReduce, FuncSum, __hip_fp8_e5m2_fnuz, NCCL_ALGO_RING, NCCL_PROTO_SIMPLE)
#endif
#if CUDART_VERSION >= 11080 && __CUDA_ARCH__ >= 900
DEFINE_ncclDevFunc(AllReduce_Sum_f8e5m2_TREE_LL, ncclFuncAllReduce, FuncSum, __hip_fp8_e5m2_fnuz, NCCL_ALGO_TREE, NCCL_PROTO_LL)
#endif
#if CUDART_VERSION >= 11080 && __CUDA_ARCH__ >= 900
DEFINE_ncclDevFunc(AllReduce_Sum_f8e5m2_TREE_LL128, ncclFuncAllReduce, FuncSum, __hip_fp8_e5m2_fnuz, NCCL_ALGO_TREE, NCCL_PROTO_LL128)
#endif
#if CUDART_VERSION >= 11080 && __CUDA_ARCH__ >= 900
DEFINE_ncclDevFunc(AllReduce_Sum_f8e5m2_TREE_SIMPLE, ncclFuncAllReduce, FuncSum, __hip_fp8_e5m2_fnuz, NCCL_ALGO_TREE, NCCL_PROTO_SIMPLE)
#endif
