#include "hip/hip_runtime.h"
#include "common.h"
#include "all_reduce.h"
#if CUDART_VERSION >= 11000 && __CUDA_ARCH__ >= 0
DEFINE_ncclDevFunc(AllReduce_MinMax_bf16_COLLNET_CHAIN_SIMPLE, ncclFuncAllReduce, FuncMinMax, __hip_bfloat16, NCCL_ALGO_COLLNET_CHAIN, NCCL_PROTO_SIMPLE)
#endif
#if CUDART_VERSION >= 11000 && __CUDA_ARCH__ >= 0
DEFINE_ncclDevFunc(AllReduce_MinMax_bf16_COLLNET_DIRECT_SIMPLE, ncclFuncAllReduce, FuncMinMax, __hip_bfloat16, NCCL_ALGO_COLLNET_DIRECT, NCCL_PROTO_SIMPLE)
#endif
#if CUDART_VERSION >= 12010 && __CUDA_ARCH__ >= 900
DEFINE_ncclDevFunc(AllReduce_MinMax_bf16_NVLS_SIMPLE, ncclFuncAllReduce, FuncMinMax, __hip_bfloat16, NCCL_ALGO_NVLS, NCCL_PROTO_SIMPLE)
#endif
#if CUDART_VERSION >= 12010 && __CUDA_ARCH__ >= 900
DEFINE_ncclDevFunc(AllReduce_MinMax_bf16_NVLS_TREE_SIMPLE, ncclFuncAllReduce, FuncMinMax, __hip_bfloat16, NCCL_ALGO_NVLS_TREE, NCCL_PROTO_SIMPLE)
#endif
#if CUDART_VERSION >= 11000 && __CUDA_ARCH__ >= 0
DEFINE_ncclDevFunc(AllReduce_MinMax_bf16_RING_LL, ncclFuncAllReduce, FuncMinMax, __hip_bfloat16, NCCL_ALGO_RING, NCCL_PROTO_LL)
#endif
#if CUDART_VERSION >= 11000 && __CUDA_ARCH__ >= 0
DEFINE_ncclDevFunc(AllReduce_MinMax_bf16_RING_LL128, ncclFuncAllReduce, FuncMinMax, __hip_bfloat16, NCCL_ALGO_RING, NCCL_PROTO_LL128)
#endif
#if CUDART_VERSION >= 11000 && __CUDA_ARCH__ >= 0
DEFINE_ncclDevFunc(AllReduce_MinMax_bf16_RING_SIMPLE, ncclFuncAllReduce, FuncMinMax, __hip_bfloat16, NCCL_ALGO_RING, NCCL_PROTO_SIMPLE)
#endif
#if CUDART_VERSION >= 11000 && __CUDA_ARCH__ >= 0
DEFINE_ncclDevFunc(AllReduce_MinMax_bf16_TREE_LL, ncclFuncAllReduce, FuncMinMax, __hip_bfloat16, NCCL_ALGO_TREE, NCCL_PROTO_LL)
#endif
#if CUDART_VERSION >= 11000 && __CUDA_ARCH__ >= 0
DEFINE_ncclDevFunc(AllReduce_MinMax_bf16_TREE_LL128, ncclFuncAllReduce, FuncMinMax, __hip_bfloat16, NCCL_ALGO_TREE, NCCL_PROTO_LL128)
#endif
#if CUDART_VERSION >= 11000 && __CUDA_ARCH__ >= 0
DEFINE_ncclDevFunc(AllReduce_MinMax_bf16_TREE_SIMPLE, ncclFuncAllReduce, FuncMinMax, __hip_bfloat16, NCCL_ALGO_TREE, NCCL_PROTO_SIMPLE)
#endif
