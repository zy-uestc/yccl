#include "hip/hip_runtime.h"
#include "common.h"
#include "all_reduce.h"
#if CUDART_VERSION >= 11080 && __CUDA_ARCH__ >= 900
DEFINE_ncclDevFunc(AllReduce_MinMax_f8e5m2_COLLNET_CHAIN_SIMPLE, ncclFuncAllReduce, FuncMinMax, __hip_fp8_e5m2_fnuz, NCCL_ALGO_COLLNET_CHAIN, NCCL_PROTO_SIMPLE)
#endif
#if CUDART_VERSION >= 11080 && __CUDA_ARCH__ >= 900
DEFINE_ncclDevFunc(AllReduce_MinMax_f8e5m2_COLLNET_DIRECT_SIMPLE, ncclFuncAllReduce, FuncMinMax, __hip_fp8_e5m2_fnuz, NCCL_ALGO_COLLNET_DIRECT, NCCL_PROTO_SIMPLE)
#endif
#if CUDART_VERSION >= 11080 && __CUDA_ARCH__ >= 900
DEFINE_ncclDevFunc(AllReduce_MinMax_f8e5m2_RING_LL, ncclFuncAllReduce, FuncMinMax, __hip_fp8_e5m2_fnuz, NCCL_ALGO_RING, NCCL_PROTO_LL)
#endif
#if CUDART_VERSION >= 11080 && __CUDA_ARCH__ >= 900
DEFINE_ncclDevFunc(AllReduce_MinMax_f8e5m2_RING_LL128, ncclFuncAllReduce, FuncMinMax, __hip_fp8_e5m2_fnuz, NCCL_ALGO_RING, NCCL_PROTO_LL128)
#endif
#if CUDART_VERSION >= 11080 && __CUDA_ARCH__ >= 900
DEFINE_ncclDevFunc(AllReduce_MinMax_f8e5m2_RING_SIMPLE, ncclFuncAllReduce, FuncMinMax, __hip_fp8_e5m2_fnuz, NCCL_ALGO_RING, NCCL_PROTO_SIMPLE)
#endif
#if CUDART_VERSION >= 11080 && __CUDA_ARCH__ >= 900
DEFINE_ncclDevFunc(AllReduce_MinMax_f8e5m2_TREE_LL, ncclFuncAllReduce, FuncMinMax, __hip_fp8_e5m2_fnuz, NCCL_ALGO_TREE, NCCL_PROTO_LL)
#endif
#if CUDART_VERSION >= 11080 && __CUDA_ARCH__ >= 900
DEFINE_ncclDevFunc(AllReduce_MinMax_f8e5m2_TREE_LL128, ncclFuncAllReduce, FuncMinMax, __hip_fp8_e5m2_fnuz, NCCL_ALGO_TREE, NCCL_PROTO_LL128)
#endif
#if CUDART_VERSION >= 11080 && __CUDA_ARCH__ >= 900
DEFINE_ncclDevFunc(AllReduce_MinMax_f8e5m2_TREE_SIMPLE, ncclFuncAllReduce, FuncMinMax, __hip_fp8_e5m2_fnuz, NCCL_ALGO_TREE, NCCL_PROTO_SIMPLE)
#endif
