#include "hip/hip_runtime.h"
#include "common.h"
#include "all_reduce.h"
#if CUDART_VERSION >= 11080 && __CUDA_ARCH__ >= 900
DEFINE_ncclDevFunc(AllReduce_Prod_f8e4m3_COLLNET_CHAIN_SIMPLE, ncclFuncAllReduce, FuncProd, __hip_fp8_e4m3_fnuz, NCCL_ALGO_COLLNET_CHAIN, NCCL_PROTO_SIMPLE)
#endif
#if CUDART_VERSION >= 11080 && __CUDA_ARCH__ >= 900
DEFINE_ncclDevFunc(AllReduce_Prod_f8e4m3_COLLNET_DIRECT_SIMPLE, ncclFuncAllReduce, FuncProd, __hip_fp8_e4m3_fnuz, NCCL_ALGO_COLLNET_DIRECT, NCCL_PROTO_SIMPLE)
#endif
#if CUDART_VERSION >= 11080 && __CUDA_ARCH__ >= 900
DEFINE_ncclDevFunc(AllReduce_Prod_f8e4m3_RING_LL, ncclFuncAllReduce, FuncProd, __hip_fp8_e4m3_fnuz, NCCL_ALGO_RING, NCCL_PROTO_LL)
#endif
#if CUDART_VERSION >= 11080 && __CUDA_ARCH__ >= 900
DEFINE_ncclDevFunc(AllReduce_Prod_f8e4m3_RING_LL128, ncclFuncAllReduce, FuncProd, __hip_fp8_e4m3_fnuz, NCCL_ALGO_RING, NCCL_PROTO_LL128)
#endif
#if CUDART_VERSION >= 11080 && __CUDA_ARCH__ >= 900
DEFINE_ncclDevFunc(AllReduce_Prod_f8e4m3_RING_SIMPLE, ncclFuncAllReduce, FuncProd, __hip_fp8_e4m3_fnuz, NCCL_ALGO_RING, NCCL_PROTO_SIMPLE)
#endif
#if CUDART_VERSION >= 11080 && __CUDA_ARCH__ >= 900
DEFINE_ncclDevFunc(AllReduce_Prod_f8e4m3_TREE_LL, ncclFuncAllReduce, FuncProd, __hip_fp8_e4m3_fnuz, NCCL_ALGO_TREE, NCCL_PROTO_LL)
#endif
#if CUDART_VERSION >= 11080 && __CUDA_ARCH__ >= 900
DEFINE_ncclDevFunc(AllReduce_Prod_f8e4m3_TREE_LL128, ncclFuncAllReduce, FuncProd, __hip_fp8_e4m3_fnuz, NCCL_ALGO_TREE, NCCL_PROTO_LL128)
#endif
#if CUDART_VERSION >= 11080 && __CUDA_ARCH__ >= 900
DEFINE_ncclDevFunc(AllReduce_Prod_f8e4m3_TREE_SIMPLE, ncclFuncAllReduce, FuncProd, __hip_fp8_e4m3_fnuz, NCCL_ALGO_TREE, NCCL_PROTO_SIMPLE)
#endif
