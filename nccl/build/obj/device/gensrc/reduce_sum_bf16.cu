#include "hip/hip_runtime.h"
#include "common.h"
#include "reduce.h"
#if CUDART_VERSION >= 11000
  #if __CUDA_ARCH__ < 0
    DEFINE_ncclDevKernel_nop(Reduce_Sum_bf16_RING_LL, ncclFuncReduce, FuncSum, __hip_bfloat16, NCCL_ALGO_RING, NCCL_PROTO_LL, 426)
  #else
    DEFINE_ncclDevKernel(Reduce_Sum_bf16_RING_LL, ncclFuncReduce, FuncSum, __hip_bfloat16, NCCL_ALGO_RING, NCCL_PROTO_LL, 426)
  #endif
#endif
#if CUDART_VERSION >= 11000 && __CUDA_ARCH__ >= 0
DEFINE_ncclDevFunc(Reduce_Sum_bf16_RING_LL, ncclFuncReduce, FuncSum, __hip_bfloat16, NCCL_ALGO_RING, NCCL_PROTO_LL)
#endif
#if CUDART_VERSION >= 11000 && __CUDA_ARCH__ >= 0
DEFINE_ncclDevFunc(Reduce_Sum_bf16_RING_LL128, ncclFuncReduce, FuncSum, __hip_bfloat16, NCCL_ALGO_RING, NCCL_PROTO_LL128)
#endif
#if CUDART_VERSION >= 11000 && __CUDA_ARCH__ >= 0
DEFINE_ncclDevFunc(Reduce_Sum_bf16_RING_SIMPLE, ncclFuncReduce, FuncSum, __hip_bfloat16, NCCL_ALGO_RING, NCCL_PROTO_SIMPLE)
#endif
