#include "hip/hip_runtime.h"
#include "common.h"
#include "reduce_scatter.h"
#if CUDART_VERSION >= 11080 && __CUDA_ARCH__ >= 900
DEFINE_ncclDevFunc(ReduceScatter_MinMax_f8e5m2_COLLNET_DIRECT_SIMPLE, ncclFuncReduceScatter, FuncMinMax, __hip_fp8_e5m2_fnuz, NCCL_ALGO_COLLNET_DIRECT, NCCL_PROTO_SIMPLE)
#endif
#if CUDART_VERSION >= 11080 && __CUDA_ARCH__ >= 900
DEFINE_ncclDevFunc(ReduceScatter_MinMax_f8e5m2_PAT_SIMPLE, ncclFuncReduceScatter, FuncMinMax, __hip_fp8_e5m2_fnuz, NCCL_ALGO_PAT, NCCL_PROTO_SIMPLE)
#endif
#if CUDART_VERSION >= 11080 && __CUDA_ARCH__ >= 900
DEFINE_ncclDevFunc(ReduceScatter_MinMax_f8e5m2_RING_LL, ncclFuncReduceScatter, FuncMinMax, __hip_fp8_e5m2_fnuz, NCCL_ALGO_RING, NCCL_PROTO_LL)
#endif
#if CUDART_VERSION >= 11080 && __CUDA_ARCH__ >= 900
DEFINE_ncclDevFunc(ReduceScatter_MinMax_f8e5m2_RING_LL128, ncclFuncReduceScatter, FuncMinMax, __hip_fp8_e5m2_fnuz, NCCL_ALGO_RING, NCCL_PROTO_LL128)
#endif
#if CUDART_VERSION >= 11080 && __CUDA_ARCH__ >= 900
DEFINE_ncclDevFunc(ReduceScatter_MinMax_f8e5m2_RING_SIMPLE, ncclFuncReduceScatter, FuncMinMax, __hip_fp8_e5m2_fnuz, NCCL_ALGO_RING, NCCL_PROTO_SIMPLE)
#endif
