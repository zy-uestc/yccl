#include "hip/hip_runtime.h"
#include "common.h"
#include "reduce_scatter.h"
#if CUDART_VERSION >= 11080 && __CUDA_ARCH__ >= 900
DEFINE_ncclDevFunc(ReduceScatter_PreMulSum_f8e4m3_COLLNET_DIRECT_SIMPLE, ncclFuncReduceScatter, FuncPreMulSum, __hip_fp8_e4m3_fnuz, NCCL_ALGO_COLLNET_DIRECT, NCCL_PROTO_SIMPLE)
#endif
#if CUDART_VERSION >= 11080 && __CUDA_ARCH__ >= 900
DEFINE_ncclDevFunc(ReduceScatter_PreMulSum_f8e4m3_PAT_SIMPLE, ncclFuncReduceScatter, FuncPreMulSum, __hip_fp8_e4m3_fnuz, NCCL_ALGO_PAT, NCCL_PROTO_SIMPLE)
#endif
#if CUDART_VERSION >= 11080 && __CUDA_ARCH__ >= 900
DEFINE_ncclDevFunc(ReduceScatter_PreMulSum_f8e4m3_RING_LL, ncclFuncReduceScatter, FuncPreMulSum, __hip_fp8_e4m3_fnuz, NCCL_ALGO_RING, NCCL_PROTO_LL)
#endif
#if CUDART_VERSION >= 11080 && __CUDA_ARCH__ >= 900
DEFINE_ncclDevFunc(ReduceScatter_PreMulSum_f8e4m3_RING_LL128, ncclFuncReduceScatter, FuncPreMulSum, __hip_fp8_e4m3_fnuz, NCCL_ALGO_RING, NCCL_PROTO_LL128)
#endif
#if CUDART_VERSION >= 11080 && __CUDA_ARCH__ >= 900
DEFINE_ncclDevFunc(ReduceScatter_PreMulSum_f8e4m3_RING_SIMPLE, ncclFuncReduceScatter, FuncPreMulSum, __hip_fp8_e4m3_fnuz, NCCL_ALGO_RING, NCCL_PROTO_SIMPLE)
#endif
