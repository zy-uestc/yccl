#include "hip/hip_runtime.h"
#include "common.h"
#include "reduce.h"
#if CUDART_VERSION >= 11000 && __CUDA_ARCH__ >= 0
DEFINE_ncclDevFunc(Reduce_PreMulSum_bf16_RING_LL, ncclFuncReduce, FuncPreMulSum, __hip_bfloat16, NCCL_ALGO_RING, NCCL_PROTO_LL)
#endif
#if CUDART_VERSION >= 11000 && __CUDA_ARCH__ >= 0
DEFINE_ncclDevFunc(Reduce_PreMulSum_bf16_RING_LL128, ncclFuncReduce, FuncPreMulSum, __hip_bfloat16, NCCL_ALGO_RING, NCCL_PROTO_LL128)
#endif
#if CUDART_VERSION >= 11000 && __CUDA_ARCH__ >= 0
DEFINE_ncclDevFunc(Reduce_PreMulSum_bf16_RING_SIMPLE, ncclFuncReduce, FuncPreMulSum, __hip_bfloat16, NCCL_ALGO_RING, NCCL_PROTO_SIMPLE)
#endif
