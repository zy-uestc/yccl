#include "hip/hip_runtime.h"
#include "common.h"
#include "reduce_scatter.h"
#if CUDART_VERSION >= 11080 && __CUDA_ARCH__ >= 900
DEFINE_ncclDevFunc(ReduceScatter_MinMax_f8e4m3_COLLNET_DIRECT_SIMPLE, ncclFuncReduceScatter, FuncMinMax, __hip_fp8_e4m3_fnuz, NCCL_ALGO_COLLNET_DIRECT, NCCL_PROTO_SIMPLE)
#endif
#if CUDART_VERSION >= 11080 && __CUDA_ARCH__ >= 900
DEFINE_ncclDevFunc(ReduceScatter_MinMax_f8e4m3_PAT_SIMPLE, ncclFuncReduceScatter, FuncMinMax, __hip_fp8_e4m3_fnuz, NCCL_ALGO_PAT, NCCL_PROTO_SIMPLE)
#endif
#if CUDART_VERSION >= 11080 && __CUDA_ARCH__ >= 900
DEFINE_ncclDevFunc(ReduceScatter_MinMax_f8e4m3_RING_LL, ncclFuncReduceScatter, FuncMinMax, __hip_fp8_e4m3_fnuz, NCCL_ALGO_RING, NCCL_PROTO_LL)
#endif
#if CUDART_VERSION >= 11080 && __CUDA_ARCH__ >= 900
DEFINE_ncclDevFunc(ReduceScatter_MinMax_f8e4m3_RING_LL128, ncclFuncReduceScatter, FuncMinMax, __hip_fp8_e4m3_fnuz, NCCL_ALGO_RING, NCCL_PROTO_LL128)
#endif
#if CUDART_VERSION >= 11080 && __CUDA_ARCH__ >= 900
DEFINE_ncclDevFunc(ReduceScatter_MinMax_f8e4m3_RING_SIMPLE, ncclFuncReduceScatter, FuncMinMax, __hip_fp8_e4m3_fnuz, NCCL_ALGO_RING, NCCL_PROTO_SIMPLE)
#endif
