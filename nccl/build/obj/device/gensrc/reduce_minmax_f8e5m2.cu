#include "hip/hip_runtime.h"
#include "common.h"
#include "reduce.h"
#if CUDART_VERSION >= 11080 && __CUDA_ARCH__ >= 900
DEFINE_ncclDevFunc(Reduce_MinMax_f8e5m2_RING_LL, ncclFuncReduce, FuncMinMax, __hip_fp8_e5m2_fnuz, NCCL_ALGO_RING, NCCL_PROTO_LL)
#endif
#if CUDART_VERSION >= 11080 && __CUDA_ARCH__ >= 900
DEFINE_ncclDevFunc(Reduce_MinMax_f8e5m2_RING_LL128, ncclFuncReduce, FuncMinMax, __hip_fp8_e5m2_fnuz, NCCL_ALGO_RING, NCCL_PROTO_LL128)
#endif
#if CUDART_VERSION >= 11080 && __CUDA_ARCH__ >= 900
DEFINE_ncclDevFunc(Reduce_MinMax_f8e5m2_RING_SIMPLE, ncclFuncReduce, FuncMinMax, __hip_fp8_e5m2_fnuz, NCCL_ALGO_RING, NCCL_PROTO_SIMPLE)
#endif
