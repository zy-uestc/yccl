#include "hip/hip_runtime.h"
#include "common.h"
#include "reduce_scatter.h"
#if CUDART_VERSION >= 11080
  #if __CUDA_ARCH__ < 900
    DEFINE_ncclDevKernel_nop(ReduceScatter_Sum_f8e5m2_RING_LL, ncclFuncReduceScatter, FuncSum, __hip_fp8_e5m2_fnuz, NCCL_ALGO_RING, NCCL_PROTO_LL, 634)
  #else
    DEFINE_ncclDevKernel(ReduceScatter_Sum_f8e5m2_RING_LL, ncclFuncReduceScatter, FuncSum, __hip_fp8_e5m2_fnuz, NCCL_ALGO_RING, NCCL_PROTO_LL, 634)
  #endif
#endif
#if CUDART_VERSION >= 11080 && __CUDA_ARCH__ >= 900
DEFINE_ncclDevFunc(ReduceScatter_Sum_f8e5m2_COLLNET_DIRECT_SIMPLE, ncclFuncReduceScatter, FuncSum, __hip_fp8_e5m2_fnuz, NCCL_ALGO_COLLNET_DIRECT, NCCL_PROTO_SIMPLE)
#endif
#if CUDART_VERSION >= 11080 && __CUDA_ARCH__ >= 900
DEFINE_ncclDevFunc(ReduceScatter_Sum_f8e5m2_PAT_SIMPLE, ncclFuncReduceScatter, FuncSum, __hip_fp8_e5m2_fnuz, NCCL_ALGO_PAT, NCCL_PROTO_SIMPLE)
#endif
#if CUDART_VERSION >= 11080 && __CUDA_ARCH__ >= 900
DEFINE_ncclDevFunc(ReduceScatter_Sum_f8e5m2_RING_LL, ncclFuncReduceScatter, FuncSum, __hip_fp8_e5m2_fnuz, NCCL_ALGO_RING, NCCL_PROTO_LL)
#endif
#if CUDART_VERSION >= 11080 && __CUDA_ARCH__ >= 900
DEFINE_ncclDevFunc(ReduceScatter_Sum_f8e5m2_RING_LL128, ncclFuncReduceScatter, FuncSum, __hip_fp8_e5m2_fnuz, NCCL_ALGO_RING, NCCL_PROTO_LL128)
#endif
#if CUDART_VERSION >= 11080 && __CUDA_ARCH__ >= 900
DEFINE_ncclDevFunc(ReduceScatter_Sum_f8e5m2_RING_SIMPLE, ncclFuncReduceScatter, FuncSum, __hip_fp8_e5m2_fnuz, NCCL_ALGO_RING, NCCL_PROTO_SIMPLE)
#endif
