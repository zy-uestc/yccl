#include "hip/hip_runtime.h"
#include "common.h"
#include "reduce_scatter.h"
#if CUDART_VERSION >= 11000 && __CUDA_ARCH__ >= 0
DEFINE_ncclDevFunc(ReduceScatter_PreMulSum_bf16_COLLNET_DIRECT_SIMPLE, ncclFuncReduceScatter, FuncPreMulSum, __hip_bfloat16, NCCL_ALGO_COLLNET_DIRECT, NCCL_PROTO_SIMPLE)
#endif
#if CUDART_VERSION >= 11000 && __CUDA_ARCH__ >= 0
DEFINE_ncclDevFunc(ReduceScatter_PreMulSum_bf16_PAT_SIMPLE, ncclFuncReduceScatter, FuncPreMulSum, __hip_bfloat16, NCCL_ALGO_PAT, NCCL_PROTO_SIMPLE)
#endif
#if CUDART_VERSION >= 11000 && __CUDA_ARCH__ >= 0
DEFINE_ncclDevFunc(ReduceScatter_PreMulSum_bf16_RING_LL, ncclFuncReduceScatter, FuncPreMulSum, __hip_bfloat16, NCCL_ALGO_RING, NCCL_PROTO_LL)
#endif
#if CUDART_VERSION >= 11000 && __CUDA_ARCH__ >= 0
DEFINE_ncclDevFunc(ReduceScatter_PreMulSum_bf16_RING_LL128, ncclFuncReduceScatter, FuncPreMulSum, __hip_bfloat16, NCCL_ALGO_RING, NCCL_PROTO_LL128)
#endif
#if CUDART_VERSION >= 11000 && __CUDA_ARCH__ >= 0
DEFINE_ncclDevFunc(ReduceScatter_PreMulSum_bf16_RING_SIMPLE, ncclFuncReduceScatter, FuncPreMulSum, __hip_bfloat16, NCCL_ALGO_RING, NCCL_PROTO_SIMPLE)
#endif
