#include "hip/hip_runtime.h"
#include "common.h"
#include "all_reduce.h"
#if CUDART_VERSION >= 11080 && __CUDA_ARCH__ >= 0
DEFINE_ncclDevFunc(AllReduce_MinMax_fp8_e4m3_COLLNET_CHAIN_SIMPLE, ncclFuncAllReduce, FuncMinMax, __hip_fp8_e4m3_fnuz, NCCL_ALGO_COLLNET_CHAIN, NCCL_PROTO_SIMPLE)
#endif
#if CUDART_VERSION >= 11080 && __CUDA_ARCH__ >= 0
DEFINE_ncclDevFunc(AllReduce_MinMax_fp8_e4m3_COLLNET_DIRECT_SIMPLE, ncclFuncAllReduce, FuncMinMax, __hip_fp8_e4m3_fnuz, NCCL_ALGO_COLLNET_DIRECT, NCCL_PROTO_SIMPLE)
#endif
#if CUDART_VERSION >= 11080 && __CUDA_ARCH__ >= 0
DEFINE_ncclDevFunc(AllReduce_MinMax_fp8_e4m3_RING_LL, ncclFuncAllReduce, FuncMinMax, __hip_fp8_e4m3_fnuz, NCCL_ALGO_RING, NCCL_PROTO_LL)
#endif
#if CUDART_VERSION >= 11080 && __CUDA_ARCH__ >= 0
DEFINE_ncclDevFunc(AllReduce_MinMax_fp8_e4m3_RING_LL128, ncclFuncAllReduce, FuncMinMax, __hip_fp8_e4m3_fnuz, NCCL_ALGO_RING, NCCL_PROTO_LL128)
#endif
#if CUDART_VERSION >= 11080 && __CUDA_ARCH__ >= 0
DEFINE_ncclDevFunc(AllReduce_MinMax_fp8_e4m3_RING_SIMPLE, ncclFuncAllReduce, FuncMinMax, __hip_fp8_e4m3_fnuz, NCCL_ALGO_RING, NCCL_PROTO_SIMPLE)
#endif
#if CUDART_VERSION >= 11080 && __CUDA_ARCH__ >= 0
DEFINE_ncclDevFunc(AllReduce_MinMax_fp8_e4m3_TREE_LL, ncclFuncAllReduce, FuncMinMax, __hip_fp8_e4m3_fnuz, NCCL_ALGO_TREE, NCCL_PROTO_LL)
#endif
#if CUDART_VERSION >= 11080 && __CUDA_ARCH__ >= 0
DEFINE_ncclDevFunc(AllReduce_MinMax_fp8_e4m3_TREE_LL128, ncclFuncAllReduce, FuncMinMax, __hip_fp8_e4m3_fnuz, NCCL_ALGO_TREE, NCCL_PROTO_LL128)
#endif
#if CUDART_VERSION >= 11080 && __CUDA_ARCH__ >= 0
DEFINE_ncclDevFunc(AllReduce_MinMax_fp8_e4m3_TREE_SIMPLE, ncclFuncAllReduce, FuncMinMax, __hip_fp8_e4m3_fnuz, NCCL_ALGO_TREE, NCCL_PROTO_SIMPLE)
#endif
