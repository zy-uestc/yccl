#include "hip/hip_runtime.h"
#include "common.h"
#include "reduce.h"
#if CUDART_VERSION >= 11080 && __CUDA_ARCH__ >= 0
DEFINE_ncclDevKernel(Reduce_Sum_fp8_e5m2_RING_LL, ncclFuncReduce, FuncSum, __hip_fp8_e5m2_fnuz, NCCL_ALGO_RING, NCCL_PROTO_LL, 465)
#endif
#if CUDART_VERSION >= 11080 && __CUDA_ARCH__ >= 0
DEFINE_ncclDevFunc(Reduce_Sum_fp8_e5m2_RING_LL, ncclFuncReduce, FuncSum, __hip_fp8_e5m2_fnuz, NCCL_ALGO_RING, NCCL_PROTO_LL)
#endif
#if CUDART_VERSION >= 11080 && __CUDA_ARCH__ >= 0
DEFINE_ncclDevFunc(Reduce_Sum_fp8_e5m2_RING_LL128, ncclFuncReduce, FuncSum, __hip_fp8_e5m2_fnuz, NCCL_ALGO_RING, NCCL_PROTO_LL128)
#endif
#if CUDART_VERSION >= 11080 && __CUDA_ARCH__ >= 0
DEFINE_ncclDevFunc(Reduce_Sum_fp8_e5m2_RING_SIMPLE, ncclFuncReduce, FuncSum, __hip_fp8_e5m2_fnuz, NCCL_ALGO_RING, NCCL_PROTO_SIMPLE)
#endif
