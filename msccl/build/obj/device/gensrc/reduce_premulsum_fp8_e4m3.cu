#include "hip/hip_runtime.h"
#include "common.h"
#include "reduce.h"
#if CUDART_VERSION >= 11080 && __CUDA_ARCH__ >= 0
DEFINE_ncclDevFunc(Reduce_PreMulSum_fp8_e4m3_RING_LL, ncclFuncReduce, FuncPreMulSum, __hip_fp8_e4m3_fnuz, NCCL_ALGO_RING, NCCL_PROTO_LL)
#endif
#if CUDART_VERSION >= 11080 && __CUDA_ARCH__ >= 0
DEFINE_ncclDevFunc(Reduce_PreMulSum_fp8_e4m3_RING_LL128, ncclFuncReduce, FuncPreMulSum, __hip_fp8_e4m3_fnuz, NCCL_ALGO_RING, NCCL_PROTO_LL128)
#endif
#if CUDART_VERSION >= 11080 && __CUDA_ARCH__ >= 0
DEFINE_ncclDevFunc(Reduce_PreMulSum_fp8_e4m3_RING_SIMPLE, ncclFuncReduce, FuncPreMulSum, __hip_fp8_e4m3_fnuz, NCCL_ALGO_RING, NCCL_PROTO_SIMPLE)
#endif
