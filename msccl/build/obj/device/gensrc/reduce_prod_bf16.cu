#include "hip/hip_runtime.h"
#include "common.h"
#include "reduce.h"
#if CUDART_VERSION >= 11000 && __CUDA_ARCH__ >= 0
DEFINE_ncclDevFunc(Reduce_Prod_bf16_RING_LL, ncclFuncReduce, FuncProd, __hip_bfloat16, NCCL_ALGO_RING, NCCL_PROTO_LL)
#endif
#if CUDART_VERSION >= 11000 && __CUDA_ARCH__ >= 0
DEFINE_ncclDevFunc(Reduce_Prod_bf16_RING_LL128, ncclFuncReduce, FuncProd, __hip_bfloat16, NCCL_ALGO_RING, NCCL_PROTO_LL128)
#endif
#if CUDART_VERSION >= 11000 && __CUDA_ARCH__ >= 0
DEFINE_ncclDevFunc(Reduce_Prod_bf16_RING_SIMPLE, ncclFuncReduce, FuncProd, __hip_bfloat16, NCCL_ALGO_RING, NCCL_PROTO_SIMPLE)
#endif
