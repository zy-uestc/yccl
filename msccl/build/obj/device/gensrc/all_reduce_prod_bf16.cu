#include "hip/hip_runtime.h"
#include "common.h"
#include "all_reduce.h"
#if CUDART_VERSION >= 11000 && __CUDA_ARCH__ >= 0
DEFINE_ncclDevFunc(AllReduce_Prod_bf16_COLLNET_CHAIN_SIMPLE, ncclFuncAllReduce, FuncProd, __hip_bfloat16, NCCL_ALGO_COLLNET_CHAIN, NCCL_PROTO_SIMPLE)
#endif
#if CUDART_VERSION >= 11000 && __CUDA_ARCH__ >= 0
DEFINE_ncclDevFunc(AllReduce_Prod_bf16_COLLNET_DIRECT_SIMPLE, ncclFuncAllReduce, FuncProd, __hip_bfloat16, NCCL_ALGO_COLLNET_DIRECT, NCCL_PROTO_SIMPLE)
#endif
#if CUDART_VERSION >= 11000 && __CUDA_ARCH__ >= 0
DEFINE_ncclDevFunc(AllReduce_Prod_bf16_RING_LL, ncclFuncAllReduce, FuncProd, __hip_bfloat16, NCCL_ALGO_RING, NCCL_PROTO_LL)
#endif
#if CUDART_VERSION >= 11000 && __CUDA_ARCH__ >= 0
DEFINE_ncclDevFunc(AllReduce_Prod_bf16_RING_LL128, ncclFuncAllReduce, FuncProd, __hip_bfloat16, NCCL_ALGO_RING, NCCL_PROTO_LL128)
#endif
#if CUDART_VERSION >= 11000 && __CUDA_ARCH__ >= 0
DEFINE_ncclDevFunc(AllReduce_Prod_bf16_RING_SIMPLE, ncclFuncAllReduce, FuncProd, __hip_bfloat16, NCCL_ALGO_RING, NCCL_PROTO_SIMPLE)
#endif
#if CUDART_VERSION >= 11000 && __CUDA_ARCH__ >= 0
DEFINE_ncclDevFunc(AllReduce_Prod_bf16_TREE_LL, ncclFuncAllReduce, FuncProd, __hip_bfloat16, NCCL_ALGO_TREE, NCCL_PROTO_LL)
#endif
#if CUDART_VERSION >= 11000 && __CUDA_ARCH__ >= 0
DEFINE_ncclDevFunc(AllReduce_Prod_bf16_TREE_LL128, ncclFuncAllReduce, FuncProd, __hip_bfloat16, NCCL_ALGO_TREE, NCCL_PROTO_LL128)
#endif
#if CUDART_VERSION >= 11000 && __CUDA_ARCH__ >= 0
DEFINE_ncclDevFunc(AllReduce_Prod_bf16_TREE_SIMPLE, ncclFuncAllReduce, FuncProd, __hip_bfloat16, NCCL_ALGO_TREE, NCCL_PROTO_SIMPLE)
#endif
